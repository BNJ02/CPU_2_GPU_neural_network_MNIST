#include "hip/hip_runtime.h"
#include "matrix.h"
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>  

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

matrix_t * alloc_matrix(unsigned rows, unsigned columns)
{
    matrix_t * res = (matrix_t*) malloc( sizeof(matrix_t) );
    // res->m = (double *) calloc(columns * rows, sizeof(double));
    res->columns = columns;
    res->rows = rows;

    size_t size = rows * columns * sizeof(double);
    hipMallocManaged(&res->m, size);       // allocation unifiée
    // hipMemset(res->m, 0, size);            // initialise à zéro (comme calloc)
    // for(int i = 0; i < size; ++i) {
    //     m[i] = 0;
    // }

    return res;
}

void destroy_matrix(matrix_t *m)
{
    //printf("free %p %p\n", m, m->m);
    hipFree(m->m);
    // free(m->m);
    free(m);
}

void print_matrix(matrix_t *m, bool is_short){
    unsigned lim_rows = 0;
    unsigned lim_col = 0;

    if (is_short)
    {
        lim_rows = MIN(m->rows, 4);
        lim_col = MIN(m->columns, 10);
    }
    else
    {
        lim_rows = m->rows;
        lim_col = m->columns;
    }

    for (int row = 0; row < lim_rows; row ++)
    {
        for (int col = 0; col < lim_col; col ++)
        {
            printf("%.2lf ", m->m[col + row * m->columns]);
        }
        if (is_short && lim_col != m->columns) printf("...");
        printf("\n");
    }
    if (is_short && lim_rows != m->rows) printf("...\n");
}

__global__
void hadamard_product_kernel(double* m1, double* m2, double* res, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        res[idx] = m1[idx] * m2[idx];
    }
}

void hadamard_product(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)   &&
             (m1->columns == res->columns)  &&
             (m1->rows == m2->rows)         &&
             (m1->rows == res->rows));

    int size = m1->rows * m1->columns;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hadamard_product_kernel<<<blocksPerGrid, threadsPerBlock>>>(m1->m, m2->m, res->m, size);
    hipDeviceSynchronize();
}

__global__
void matrix_sum_kernel(double* m1, double* m2, double* res, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        res[idx] = m1[idx] + m2[idx];
    }
}

void matrix_sum(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));

    int size = m1->rows * m1->columns;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    matrix_sum_kernel<<<blocksPerGrid, threadsPerBlock>>>(m1->m, m2->m, res->m, size);
    hipDeviceSynchronize();
}

__global__
void matrix_minus_kernel(double* m1, double* m2, double* res, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        res[idx] = m1[idx] - m2[idx];
    }
}

void matrix_minus(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert ( (m1->columns == m2->columns)  &&
             (m1->columns == res->columns) &&
             (m1->rows == m2->rows)        &&
             (m1->rows == res->rows));
             
    int size = m1->rows * m1->columns;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    matrix_minus_kernel<<<blocksPerGrid, threadsPerBlock>>>(m1->m, m2->m, res->m, size);
    hipDeviceSynchronize();
}

__global__
void matrix_dot_kernel(const double *A, const double *B, double *C,
                       int A_rows, int A_cols, int B_cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        double acc = 0.0;
        for (int k = 0; k < A_cols; ++k) {
            acc += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = acc;
    }
}

void matrix_dot(matrix_t *m1, matrix_t *m2, matrix_t *res)
{
    assert((m1->columns == m2->rows) &&
           (m1->rows == res->rows) &&
           (m2->columns == res->columns));

    dim3 blockDim(16, 16);
    dim3 gridDim((m2->columns + 15) / 16, (m1->rows + 15) / 16);

    matrix_dot_kernel<<<gridDim, blockDim>>>(m1->m, m2->m, res->m,
        m1->rows, m1->columns, m2->columns);

    hipDeviceSynchronize(); // attendre que le résultat soit prêt
}

__device__ double sigmoid_device(double x) {
    return 1.0 / (1.0 + exp(-x));
}

__global__
void matrix_sigmoid_kernel(double* m, double* res, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        res[idx] = sigmoid_device(m[idx]);
    }
}

void matrix_function(matrix_t *m1, double (*f)(double), matrix_t *res)
{
    assert ( (m1->columns == res->columns) &&             
             (m1->rows == res->rows));

    int size = m1->rows * m1->columns;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    matrix_sigmoid_kernel<<<blocksPerGrid, threadsPerBlock>>>(m1->m, res->m, size);
    hipDeviceSynchronize();
}

void matrix_transpose(matrix_t *m1, matrix_t *res)
{
    assert ( (m1->columns == res->rows) &&             
             (m1->rows == res->columns));
    
    for (int row = 0; row < m1->rows; row++)
    {
        for (int col = 0; col < m1->columns; col ++)
        {
            res->m[row + col * m1->rows] = m1->m[col + row * m1->columns];
        }
    }
}

__global__
void matrix_scalar_kernel(double* m, double scalar, double* res, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        res[idx] = m[idx] * scalar;
    }
}

void matrix_scalar(matrix_t *m1, double s, matrix_t *res)
{
    assert ( (m1->rows == res->rows) &&             
             (m1->columns == res->columns));

    int size = m1->rows * m1->columns;
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    matrix_scalar_kernel<<<blocksPerGrid, threadsPerBlock>>>(m1->m, s, res->m, size);
    hipDeviceSynchronize();
}

void matrix_memcpy(matrix_t *dest, const matrix_t *src)
{
    assert ( (dest->rows == src->rows)      &&             
             (dest->columns == src->columns));

    memcpy(dest->m, src->m, src->columns * src->rows * sizeof(double));     
}