#include "hip/hip_runtime.h"
#include "ann.h"
#include "matrix.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <float.h>
#include <stdbool.h>
#include <stdint.h>

double normalRand(double mu, double sigma);
void init_weight(matrix_t* w, unsigned nneurones_prev);
void print_layer(layer_t *layer);

double normalRand(double mu, double sigma)
{
	const double epsilon = DBL_MIN;
	const double two_pi = 2.0*M_PI;
    static bool generate = false;
    static double z1;

	generate = !generate;

	if (!generate)
	   return z1 * sigma + mu;

	double u1, u2;
	do
	 {
	   u1 = (double) rand() / RAND_MAX;
	   u2 = (double) rand() / RAND_MAX;
	 }
	while ( u1 <= epsilon );

	double z0;
	z0 = sqrt(-2.0 * log(u1)) * cos(two_pi * u2);
	z1 = sqrt(-2.0 * log(u1)) * sin(two_pi * u2);
	return z0 * sigma + mu;
}

void init_weight(matrix_t* w, unsigned nneurones_prev)
{
    for (int idx = 0; idx < w->columns * w->rows; idx ++)
    {
        w->m[idx] = normalRand(0, 1 / sqrt(nneurones_prev));
    }
}

ann_t * create_ann(double alpha, unsigned minibatch_size, unsigned number_of_layers, unsigned* nneurons_per_layer)
{
    ann_t * nn = (ann_t *)malloc(sizeof(ann_t));

    nn->layers = (layer_t **)malloc(number_of_layers * sizeof(layer_t *));
    nn->number_of_layers = number_of_layers;
    nn->alpha = alpha;
    nn->minibatch_size = minibatch_size;

    nn->layers[0] = create_layer(0, nneurons_per_layer[0], minibatch_size, minibatch_size);
    for (int l = 1; l < number_of_layers; l++)
    {
        nn->layers[l] = create_layer(l, nneurons_per_layer[l], nneurons_per_layer[l-1], minibatch_size);
    }

    return nn;
}

layer_t * create_layer(unsigned layer_number, unsigned number_of_neurons, unsigned nneurons_previous_layer, unsigned minibatch_size)
{
    layer_t * layer = (layer_t*) malloc(sizeof(layer_t));

    layer->number_of_neurons = number_of_neurons;
    layer->minibatch_size = minibatch_size;    
    layer->activations = alloc_matrix(number_of_neurons, minibatch_size);
    layer->z = alloc_matrix(number_of_neurons, minibatch_size);
    layer->delta = alloc_matrix(number_of_neurons, minibatch_size);
    layer->weights = alloc_matrix(number_of_neurons, nneurons_previous_layer);    
    layer->biases = alloc_matrix(number_of_neurons, 1);

    if (layer_number > 0)
    {
        init_weight(layer->weights, nneurons_previous_layer);
    }

    return layer;
}

void set_input(ann_t *nn, matrix_t* input){
    matrix_memcpy(nn->layers[0]->activations, input);
}

void print_layer(layer_t *layer)
{
    printf("-- neurons:%d, minibatch size:%d\n", layer->number_of_neurons, layer->minibatch_size);

    printf(">> Weighted inputs --\n");
    print_matrix(layer->z, true);
    printf(">> Activations --\n");
    print_matrix(layer->activations, true);
    
    printf(">> Weights --\n");
    print_matrix(layer->weights, true);
    printf(">> Biases --\n");
    print_matrix(layer->biases, true);

    printf(">> Delta --\n");
    print_matrix(layer->delta, true);
    
}

void print_nn(ann_t *nn)
{
    printf("ANN -- nlayers:%d, alpha:%lf, minibatch size: %d\n", nn->number_of_layers, nn->alpha, nn->minibatch_size);
    for (int l = 0; l < nn->number_of_layers; l++)
    {
        printf("Layer %d ", l);
        print_layer(nn->layers[l]);
    }
}

/* ============================================================================
* Forward propagation
*   ‑ persistent scratch:
*       z1[l]  : W_l * a_{l‑1}
*       z2[l]  : b_l * 1
*       one    : vector of 1 used for bias broadcast
* ============================================================================
*/
void forward(ann_t *nn) {
    /* persistent scratch : one, z1, z2 ------------------------------------ */
    static matrix_t **z1 = nullptr, **z2 = nullptr;
    static matrix_t  *one1 = nullptr;            // 1 × m (bias broadcast)
    static unsigned   lay_cached = 0, mb_cached = 0;

    if (lay_cached != nn->number_of_layers || mb_cached != nn->minibatch_size)
    {
        /* free previous */
        if (z1)
        {
            for (unsigned l = 1; l < lay_cached; ++l) {
                destroy_matrix(z1[l]);
                destroy_matrix(z2[l]);
            }
            free(z1); free(z2); destroy_matrix(one1);
        }

        lay_cached = nn->number_of_layers;
        mb_cached  = nn->minibatch_size;

        z1  = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        z2  = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));

        for (unsigned l = 1; l < lay_cached; ++l) {
            z1[l] = alloc_matrix(nn->layers[l]->number_of_neurons, mb_cached);
            z2[l] = alloc_matrix(nn->layers[l]->number_of_neurons, mb_cached);
        }
        one1 = alloc_matrix(1, mb_cached);
        for (unsigned i = 0; i < one1->rows * one1->columns; ++i) one1->m[i] = 1.0;
    }

    /* actual propagation --------------------------------------------------- */
    for (unsigned l = 1; l < nn->number_of_layers; ++l) {
        matrix_dot(nn->layers[l]->weights, nn->layers[l-1]->activations, z1[l]);
        matrix_dot(nn->layers[l]->biases,  one1,                         z2[l]);
        matrix_sum(z1[l], z2[l], nn->layers[l]->z);
        matrix_function(nn->layers[l]->z, nn->layers[l]->activations, false);
    }
}
 
 
/* ============================================================================
* Back‑propagation
*   ‑ persistent scratch:
*       tw[l]        : transposed weights / grad_w
*       delta_tmp[l] : temporary δ before Hadamard
*       dfz[l]       : f'(z_{l})
*       b1[l]        : grad_b
*       one          : vector of 1 for bias reduction
* ============================================================================
*/
void backward(ann_t *nn, matrix_t *y)
{
    const unsigned L = nn->number_of_layers - 1;

    /* persistent scratch ---------------------------------------------------- */
    static matrix_t **tw = nullptr, **delta_tmp = nullptr,
            **dfz = nullptr, **w1 = nullptr, **ta = nullptr,
            **b1 = nullptr;                    // ∇b
    static matrix_t  *one2 = nullptr;                  // m × 1 (bias reduce)
    static unsigned   lay_cached = 0, mb_cached = 0;

    if (lay_cached != nn->number_of_layers || mb_cached != nn->minibatch_size)
    {
        /* free previous */
        if (tw) {
            for (unsigned l = 1; l < lay_cached; ++l) {
                destroy_matrix(tw[l]); destroy_matrix(delta_tmp[l]);
                destroy_matrix(dfz[l]); destroy_matrix(w1[l]);
                destroy_matrix(ta[l]); destroy_matrix(b1[l]);
            }

            free(tw); free(delta_tmp); free(dfz);
            free(w1); free(ta); free(b1); destroy_matrix(one2);
        }

        lay_cached = nn->number_of_layers;
        mb_cached  = nn->minibatch_size;

        tw        = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        delta_tmp = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        dfz       = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        w1        = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        ta        = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));
        b1        = (matrix_t**)malloc(lay_cached * sizeof(matrix_t*));

        for (unsigned l = 1; l < lay_cached; ++l) {
            tw[l]        = alloc_matrix(nn->layers[l-1]->number_of_neurons,
                                        nn->layers[l]->number_of_neurons);
            delta_tmp[l] = alloc_matrix(nn->layers[l-1]->number_of_neurons,
                                        mb_cached);
            dfz[l]       = alloc_matrix(nn->layers[l]->number_of_neurons,
                                        mb_cached);
            w1[l]        = alloc_matrix(nn->layers[l]->number_of_neurons,
                                        nn->layers[l-1]->number_of_neurons);
            ta[l]        = alloc_matrix(mb_cached,
                                        nn->layers[l-1]->number_of_neurons);
            b1[l]        = alloc_matrix(nn->layers[l]->number_of_neurons, 1);
        }
        /* dfz[0] never used but keep consistent for simplicity */
        dfz[0] = alloc_matrix(nn->layers[0]->number_of_neurons, mb_cached);

        one2 = alloc_matrix(mb_cached, 1);
        for (unsigned i = 0; i < one2->rows * one2->columns; ++i) one2->m[i] = 1.0;
    }

    /* --- δ^L -------------------------------------------------------------- */
    matrix_minus(nn->layers[L]->activations, y, nn->layers[L]->delta);
    matrix_function(nn->layers[L]->z, dfz[L], true);
    hadamard_product(nn->layers[L]->delta, dfz[L], nn->layers[L]->delta);

    /* --- layers L .. 1 ---------------------------------------------------- */
    for (int l = L; l > 0; --l) {
        /* ∇Wᶫ : delta^l × (a^{l-1})ᵀ   ------------------------------------ */
        matrix_transpose(nn->layers[l-1]->activations, ta[l]);
        matrix_dot(nn->layers[l]->delta, ta[l], w1[l]);
        matrix_scalar(w1[l], nn->alpha / mb_cached, w1[l]);
        matrix_minus(nn->layers[l]->weights, w1[l], nn->layers[l]->weights);

        /* ∇bᶫ : delta^l × 1 ---------------------------------------------- */
        matrix_dot(nn->layers[l]->delta, one2, b1[l]);
        matrix_scalar(b1[l], nn->alpha / mb_cached, b1[l]);
        matrix_minus(nn->layers[l]->biases, b1[l], nn->layers[l]->biases);

        if (l > 1) {
            /* δ^{l-1} ------------------------------------------------------ */
            matrix_transpose(nn->layers[l]->weights, tw[l]);
            matrix_dot(tw[l], nn->layers[l]->delta, delta_tmp[l]);
            matrix_function(nn->layers[l-1]->z, dfz[l-1], true);
            hadamard_product(delta_tmp[l], dfz[l-1], nn->layers[l-1]->delta);
        }
    }
}

 